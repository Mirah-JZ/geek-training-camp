
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

__global__ void reduce3(float *g_idata, float *g_odata) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) { sdata[tid] += sdata[tid + s]; }
        __syncthreads();
    }
}

int main(void) {
    int N = 100000000;
    float *g_indata_host, *g_indata_device, *g_outdata_host, *g_outdata_device;
    g_indata_host = (float *) malloc(N * sizeof(float));
    g_outdata_host = (float *) malloc(sizeof(float));

    hipMalloc(&g_indata_device, N * sizeof(float));
    hipMalloc(&g_outdata_device, sizeof(float));

    for (auto i = 0; i < N; i++) {
        g_indata_host[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);;
    }

    hipMemcpy(g_indata_device, g_indata_host, N * sizeof(float), hipMemcpyHostToDevice);

//    This is where the code is run
    auto start = high_resolution_clock::now();
    reduce3<<<(N + 255) / 256, 256>>>(g_indata_device, g_outdata_device);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << "Time taken by function: "
              << duration.count() << " microseconds" << std::endl;
    hipFree(g_indata_device);
    hipFree(g_outdata_device);
    free(g_indata_host);
    free(g_outdata_host);

}